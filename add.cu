// add_gpu.cpp -> run the add.cpp code on a GPU


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
using namespace std;

// function to add elements of two arrays
/*
GPU can run a function called a `kernel` in CUDA.
Add the specifier __global__ to the function, it tells
the CUDA C++ compiler that the function runs on the GPU
and can be called from CPU code.
* __global__ functions are also known as kernels
Code running on GPU -> device code
             on CPU -> host code
*/
__global__
void add(int n, float *x, float *y){
    for (int i=0; i<n; i++){
        y[i] = x[i] + y[i];
    }
}

int main(){
    int N = 1<<20;      // 1M elements

    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for(int i=0; i<N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the CPU
    /*
    launch one GPU thread to run add().
    */
    add<<<1, 1>>>(N, x, y);

    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;

    for(int i=0; i<N; i++){
        // calculate maximum absolute difference between an 
        // element of array y and the value 3.0
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    cout << "Max Error: " << maxError << endl;

    // free memory
    hipFree(x);
    hipFree(y);

    return 0;
}